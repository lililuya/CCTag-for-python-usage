/*
 * Copyright 2016, Simula Research Laboratory
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at http://mozilla.org/MPL/2.0/.
 */
#include "keep_time.hpp"
#include <stdio.h>
#include "debug_macros.hpp"

// #include <assert.h>

using namespace std;

namespace cctag {

KeepTime::KeepTime( hipStream_t s )
    : _stream( s )
{
    hipEventCreate( &_start );
    hipEventCreate( &_stop );
}

KeepTime::~KeepTime( )
{
    hipEvent_t ev;
    hipError_t err;
    while( !_other_events.empty() ) {
        ev = _other_events.front();
        _other_events.pop_front();
        err = hipEventSynchronize( ev );
        POP_CUDA_FATAL_TEST( err, "Couldn't wait for other event in ~KeepTime: " );
        err = hipEventDestroy( ev );
        POP_CUDA_FATAL_TEST( err, "Couldn't destroy other event in ~KeepTime: " );
    }

    hipEventDestroy( _start );
    hipEventDestroy( _stop );
}

void KeepTime::start()
{
    hipEventRecord( _start, _stream );
}

void KeepTime::stop( )
{
    hipEventRecord( _stop, _stream );
}

void KeepTime::report( const char* msg )
{
    hipEventSynchronize( _stop );
    float diff;
    hipEventElapsedTime( &diff, _start, _stop );
    fprintf(stderr,"%s %f ms\n", msg, diff );
}

float KeepTime::getElapsed( )
{
    hipEventSynchronize( _stop );
    float diff;
    hipEventElapsedTime( &diff, _start, _stop );
    return diff;
}

void KeepTime::waitFor( hipStream_t otherStream )
{
    hipEvent_t ev;
    hipError_t err = hipEventCreate( &ev );
    POP_CUDA_FATAL_TEST( err, "Couldn't create sync event in KeepTime: " );

    _other_events.push_back( ev );

    err = hipEventRecord( ev, otherStream );
    POP_CUDA_FATAL_TEST( err, "Couldn't insert event into other stream in KeepTime: " );

    err = hipStreamWaitEvent( _stream, ev, 0 );
    POP_CUDA_FATAL_TEST( err, "Couldn't synchronize on event from other stream in KeepTime: " );

}

} // namespace cctag

