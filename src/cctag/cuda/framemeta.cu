/*
 * Copyright 2016, Simula Research Laboratory
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at http://mozilla.org/MPL/2.0/.
 */
#include <iostream>

#include "onoff.h"
#include "framemeta.h"
#include "debug_macros.hpp"

namespace cctag {

/*************************************************************
 * FrameMeta
 * This structure is required for moving things such as counters
 * between host and device. It is meant to replace a memory
 * mapping function that is unreliable.
 */
struct FrameMeta
{
    int   hysteresis_block_counter;
    int   connect_component_block_counter;
    int   ring_counter;
    int   ring_counter_max;
    float identification_result;
    int   identification_resct;
#ifndef NDEBUG
    int   num_edges_thinned;
#endif // NDEBUG
    int   list_size_all_edgecoords;
    int   list_size_voters;
    int   list_size_chosen_idx;
    int   list_size_inner_points;
    int   list_size_interm_inner_points;
    int   swap_buffers_after_sort;
};

__device__
FrameMeta frame_meta[ FRAME_META_MAX_PIPES * FRAME_META_MAX_LEVELS ];

__host__
FrameMetaPtr::FrameMetaPtr( int pipeId, int frameId )
    : _pipeId( pipeId )
    , _frameId( frameId )
{
    if( pipeId >= FRAME_META_MAX_PIPES ) {
	std::cerr << __FILE__ << ":" << __LINE__ << std::endl
		  << "Requesting more than " << FRAME_META_MAX_PIPES << " CUDA pipelines."
		  << std::endl
		  << "This requires a recompile."
		  << std::endl;
        exit( -1 );
    }
    if( frameId >= FRAME_META_MAX_LEVELS ) {
	std::cerr << __FILE__ << ":" << __LINE__ << std::endl
		  << "Requesting more than " << FRAME_META_MAX_LEVELS << " CUDA pipelines."
		  << std::endl
		  << "This requires a recompile."
		  << std::endl;
        exit( -1 );
    }

    hipError_t err;
    err = hipGetSymbolAddress( &_d_symbol_ptr, HIP_SYMBOL(frame_meta ));
    POP_CUDA_FATAL_TEST( err, "Could not recover the symbol address for FrameMetas" );
}

#define HOST_DEVICE_TRANSFER_CASE( cond, val ) \
    case cond: \
        offset = (intptr_t)&frame_meta[my_meta].val - (intptr_t)frame_meta; \
        break;

#define HOST_DEVICE_TRANSFER_ALL_CASES \
    HOST_DEVICE_TRANSFER_CASE( Hysteresis_block_counter, hysteresis_block_counter ) \
    HOST_DEVICE_TRANSFER_CASE( Connect_component_block_counter, connect_component_block_counter ) \
    HOST_DEVICE_TRANSFER_CASE( Ring_counter, ring_counter ) \
    HOST_DEVICE_TRANSFER_CASE( Ring_counter_max, ring_counter_max ) \
    HOST_DEVICE_TRANSFER_CASE( Identification_resct, identification_resct ) \
    HOST_DEVICE_TRANSFER_CASE( List_size_all_edgecoords, list_size_all_edgecoords ) \
    HOST_DEVICE_TRANSFER_CASE( List_size_voters, list_size_voters ) \
    HOST_DEVICE_TRANSFER_CASE( List_size_chosen_idx, list_size_chosen_idx ) \
    HOST_DEVICE_TRANSFER_CASE( List_size_inner_points, list_size_inner_points ) \
    HOST_DEVICE_TRANSFER_CASE( List_size_interm_inner_points, list_size_interm_inner_points ) \
    HOST_DEVICE_TRANSFER_CASE( Swap_buffers_after_sort, swap_buffers_after_sort )

__host__
void FrameMetaPtr::toDevice( FrameMetaEnum e, int val, hipStream_t stream )
{
    POP_CHK_CALL_IFSYNC;
    const size_t my_meta = _pipeId*FRAME_META_MAX_LEVELS+_frameId;
    intptr_t offset;
    switch( e ) {
    HOST_DEVICE_TRANSFER_ALL_CASES
#ifndef NDEBUG
    HOST_DEVICE_TRANSFER_CASE( Num_edges_thinned, num_edges_thinned )
#endif // NDEBUG

    case Identification_result:
    	std::cerr << __FILE__ << ":" << __LINE__ << std::endl
		  << __FUNCTION__ << std::endl
		  << "Trying to copy an int to FrameMeta::<float>" << std::endl
		  << "Type is incorrect." << std::endl;
	exit( -1 );
    default :
    	std::cerr << __FILE__ << ":" << __LINE__ << std::endl
		  << __FUNCTION__ << std::endl
		  << "Trying to copy an unknown FrameMeta element." << std::endl;
	exit( -1 );
    }
    hipError_t err;
    err = hipMemcpyToSymbolAsync(HIP_SYMBOL( frame_meta), // _d_symbol_ptr,
                                   &val,
                                   sizeof(int),
                                   offset,
                                   hipMemcpyHostToDevice,
                                   stream );
    POP_CHK_CALL_IFSYNC;
    POP_CUDA_FATAL_TEST( err, "Could not copy int variable to device symbol: " );
}

__host__
void FrameMetaPtr::toDevice_D2S( FrameMetaEnum e, int* val, hipStream_t stream )
{
    POP_CHK_CALL_IFSYNC;
    const size_t my_meta = _pipeId*FRAME_META_MAX_LEVELS+_frameId;
    intptr_t offset;
    switch( e ) {
    HOST_DEVICE_TRANSFER_ALL_CASES
#ifndef NDEBUG
    HOST_DEVICE_TRANSFER_CASE( Num_edges_thinned, num_edges_thinned )
#endif // NDEBUG

    case Identification_result:
    	std::cerr << __FILE__ << ":" << __LINE__ << std::endl
		  << __FUNCTION__ << std::endl
		  << "Trying to copy an int to FrameMeta::<float>" << std::endl
		  << "Type is incorrect." << std::endl;
        exit( -1 );
    default :
    	std::cerr << __FILE__ << ":" << __LINE__ << std::endl
		  << __FUNCTION__ << std::endl
		  << "Trying to copy an unknown FrameMeta element." << std::endl;
        exit( -1 );
    }
    hipError_t err;
    err = hipMemcpyToSymbolAsync(HIP_SYMBOL( frame_meta), // _d_symbol_ptr,
                                   val,
                                   sizeof(int),
                                   offset,
                                   hipMemcpyDeviceToDevice,
                                   stream );
    POP_CHK_CALL_IFSYNC;
    POP_CUDA_FATAL_TEST( err, "Could not copy int variable to device symbol" );
}

__host__
void FrameMetaPtr::toDevice( FrameMetaEnum e, float val, hipStream_t stream )
{
    POP_CHK_CALL_IFSYNC;
    const size_t my_meta = _pipeId*FRAME_META_MAX_LEVELS+_frameId;
    intptr_t offset;
    switch( e ) {
    HOST_DEVICE_TRANSFER_CASE( Identification_result, identification_result )
    default :
    	std::cerr << __FILE__ << ":" << __LINE__ << std::endl
		  << __FUNCTION__ << std::endl
		  << "Trying to copy an unknown FrameMeta element." << std::endl;
        exit( -1 );
    }
    hipError_t err;
    err = hipMemcpyToSymbolAsync(HIP_SYMBOL( frame_meta), // _d_symbol_ptr,
                                   &val,
                                   sizeof(float),
                                   offset,
                                   hipMemcpyHostToDevice,
                                   stream );
    POP_CHK_CALL_IFSYNC;
    POP_CUDA_FATAL_TEST( err, "Could not copy float variable to device symbol" );
}

__host__
void FrameMetaPtr::toDevice_D2S( FrameMetaEnum e, float* val, hipStream_t stream )
{
    POP_CHK_CALL_IFSYNC;
    const size_t my_meta = _pipeId*FRAME_META_MAX_LEVELS+_frameId;
    intptr_t offset;
    switch( e ) {
    HOST_DEVICE_TRANSFER_CASE( Identification_result, identification_result )
    default :
    	std::cerr << __FILE__ << ":" << __LINE__ << std::endl
		  << __FUNCTION__ << std::endl
		  << "Trying to copy an unknown FrameMeta element." << std::endl;
        exit( -1 );
    }
    hipError_t err;
    err = hipMemcpyToSymbolAsync(HIP_SYMBOL( frame_meta), // _d_symbol_ptr,
                                   val,
                                   sizeof(float),
                                   offset,
                                   hipMemcpyDeviceToDevice,
                                   stream );
    POP_CHK_CALL_IFSYNC;
    POP_CUDA_FATAL_TEST( err, "Could not copy float variable to device symbol" );
}

__host__
void FrameMetaPtr::fromDevice( FrameMetaEnum e, int& val, hipStream_t stream )
{
    POP_CHK_CALL_IFSYNC;
    const size_t my_meta = _pipeId*FRAME_META_MAX_LEVELS+_frameId;
    intptr_t offset;
    switch( e ) {
    HOST_DEVICE_TRANSFER_ALL_CASES
#ifndef NDEBUG
    HOST_DEVICE_TRANSFER_CASE( Num_edges_thinned, num_edges_thinned )
#endif // NDEBUG
    case Identification_result:
    	std::cerr << __FILE__ << ":" << __LINE__ << std::endl
		  << __FUNCTION__ << std::endl
		  << "Trying to fetch an int to FrameMeta::<float>" << std::endl
		  << "Type is incorrect." << std::endl;
        exit( -1 );
    default :
    	std::cerr << __FILE__ << ":" << __LINE__ << std::endl
		  << __FUNCTION__ << std::endl
		  << "Trying to fetch an unknown FrameMeta element." << std::endl;
        exit( -1 );
    }
    hipError_t err;
    err = hipMemcpyFromSymbolAsync( &val,
                                     HIP_SYMBOL(frame_meta), // _d_symbol_ptr,
                                     sizeof(int),
                                     offset,
                                     hipMemcpyDeviceToHost,
                                     stream );
    POP_CHK_CALL_IFSYNC;
    POP_CUDA_FATAL_TEST( err, "Could not copy int variable from device symbol: " );
}

__host__
void FrameMetaPtr::fromDevice( FrameMetaEnum e, float& val, hipStream_t stream )
{
    POP_CHK_CALL_IFSYNC;
    const size_t my_meta = _pipeId*FRAME_META_MAX_LEVELS+_frameId;
    intptr_t offset;
    switch( e ) {
    HOST_DEVICE_TRANSFER_CASE( Identification_result, identification_result )
    default :
    	std::cerr << __FILE__ << ":" << __LINE__ << std::endl
		  << __FUNCTION__ << std::endl
		  << "Trying to fetch an unknown FrameMeta element." << std::endl;
        exit( -1 );
    }
    hipError_t err;
    err = hipMemcpyFromSymbolAsync( &val,
                                     HIP_SYMBOL(frame_meta), // _d_symbol_ptr,
                                     sizeof(float),
                                     offset,
                                     hipMemcpyDeviceToHost,
                                     stream );
    POP_CHK_CALL_IFSYNC;
    POP_CUDA_FATAL_TEST( err, "Could not copy float variable from device symbol: " );
}

#define OFFSET_GETTER_BODY( type, name ) \
    __device__ \
    type& FrameMetaPtr::name() { \
        const size_t my_meta = _pipeId*FRAME_META_MAX_LEVELS+_frameId; \
        return frame_meta[my_meta].name; \
    } \
    \
    __device__ \
    const type& FrameMetaPtr::name() const { \
        const size_t my_meta = _pipeId*FRAME_META_MAX_LEVELS+_frameId; \
        return frame_meta[my_meta].name; \
    }


OFFSET_GETTER_BODY( int,   hysteresis_block_counter )
OFFSET_GETTER_BODY( int,   connect_component_block_counter )
OFFSET_GETTER_BODY( int,   ring_counter )
OFFSET_GETTER_BODY( int,   ring_counter_max )
OFFSET_GETTER_BODY( float, identification_result )
OFFSET_GETTER_BODY( int,   identification_resct )
#ifndef NDEBUG
OFFSET_GETTER_BODY( int,   num_edges_thinned )
#endif // NDEBUG
OFFSET_GETTER_BODY( int,   list_size_all_edgecoords )
OFFSET_GETTER_BODY( int,   list_size_voters )
OFFSET_GETTER_BODY( int,   list_size_chosen_idx )
OFFSET_GETTER_BODY( int,   list_size_inner_points )
OFFSET_GETTER_BODY( int,   list_size_interm_inner_points )
OFFSET_GETTER_BODY( int,   swap_buffers_after_sort )

}; // namespace cctag

